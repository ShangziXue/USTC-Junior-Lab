
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#include "math.h"
#define BLOCK_SIZE 32
int block_size;


void generate_matrix(float *mat, int a, int b)
{
	int i, j, m, n;
	m = a * block_size;
	n = b * block_size;

	for(i = 0; i < m; i++)
		for(j = 0; j < n; j++)
			mat[i * n + j] = (float)(rand() % 100) / 10.0;
}

void serial_compute(float *A, float *B, float *C)
{
	int i, j, k, m, n;
	m = 10 * block_size;
	n = 20 * block_size;

	for(i = 0; i < m; i++)
		for(j = 0;j < n; j++)
		{
			float sum = 0;
			for(k = 0; k < m; k++)
				sum += A[i * m + k] * B[k * n + j];
			C[i * n + j] = sum;
		}
}

bool check_result(float *C1, float *C2)
{
	int m = 10 * block_size, n = 20 * block_size;
	for(int i = 0; i < m; i++)
		for(int j = 0; j < n; j++)
			if(fabs(C1[i * n + j] - C2[i * n + j]) > 1e-2)
			{
				printf("C1:%lf, C2:%lf\n", C1[i * n + j], C2[i * n + j]);
				return false;
			}

	return true;
}

__global__ void tiled_parallel_compute(float *A, float *B, float *C, int block_size)
{
	__shared__ float block_A[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ float block_B[BLOCK_SIZE][BLOCK_SIZE];

	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int column = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int m = 10 * block_size, n = 20 * block_size;
	int t_x = threadIdx.x, t_y = threadIdx.y;
	float sum = 0;

	for(int i = 0; i < m / BLOCK_SIZE; i++)
	{
		block_A[t_y][t_x] = A[row * m + i * BLOCK_SIZE + t_x];
		block_B[t_y][t_x] = B[(i * BLOCK_SIZE + t_y) * n + column];
		__syncthreads();

		for(int j = 0; j < BLOCK_SIZE; j++)
			sum += block_A[t_y][j] * block_B[j][t_x];
		__syncthreads();
	}

	C[row * n + column] = sum;

}

__global__ void parallel_compute(float *A, float *B, float *C, int block_size)
{
	int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int column = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int m = 10 * block_size, n = 20 * block_size;
	float sum = 0;

	if(row < m && column < n)
	{
		for(int i = 0; i < m; i++)
			sum += A[row * m + i] * B[i * n + column];
		C[row * n + column] = sum;
	}

}

int main(int argc, char const *argv[])
{

	float *A, *B, *C1, *C2, *CUDA_A, *CUDA_B, *CUDA_C;
	clock_t start, end;
	double t0, t1, t2;
	int m, n;

	block_size = atoi(argv[1]) / BLOCK_SIZE * BLOCK_SIZE;
	m = 10 * block_size;
	n = 20 * block_size;
	A = (float*)malloc(sizeof(float) * m * m);
	B = (float*)malloc(sizeof(float) * m * n);
	C1 = (float*)malloc(sizeof(float) * m * n);
	C2 = (float*)malloc(sizeof(float) * m * n);
	srand(time(NULL));

	generate_matrix(A, 10, 10);
	generate_matrix(B, 10, 20);

	start = clock();
	serial_compute(A, B, C1);
	end = clock();
	t0 = (((double) (end - start)) / CLOCKS_PER_SEC );
	printf("Serial algorithm costs %lfs.\n", t0);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 blocks(n / BLOCK_SIZE, m / BLOCK_SIZE, 1);
	hipMalloc(&CUDA_A, sizeof(float) * m * m);
	hipMalloc(&CUDA_B, sizeof(float) * m * n);
	hipMalloc(&CUDA_C, sizeof(float) * m * n);
	// Copy to GPU memory
	hipMemcpy(CUDA_A, A, m * m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(CUDA_B, B, m * n * sizeof(float), hipMemcpyHostToDevice);
	// parallel
	start = clock();
	parallel_compute<<<blocks, threads>>>(CUDA_A, CUDA_B, CUDA_C, block_size);
	end = clock();
	hipMemcpy(C2, CUDA_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
	t1 = ((double) (clock() - start)) / CLOCKS_PER_SEC;

	bool right = check_result(C1, C2);
	if(right)
		printf("Parallel algorithm costs %lfs, speedup is %lf, result is right.\n", t1, t0 / t1 / 10);
	else
		printf("Result is wrong.\n");

	// tiled
	start = clock();
	tiled_parallel_compute<<<blocks, threads>>>(CUDA_A, CUDA_B, CUDA_C, block_size);
	end = clock();
	hipMemcpy(C2, CUDA_C, m * n * sizeof(float), hipMemcpyDeviceToHost);
	t2 = ((double) (clock() - start)) / CLOCKS_PER_SEC;

	right = check_result(C1, C2);
	if(right)
		printf("Parallel tiled algorithm costs %lfs, speedup is %lf, result is right.\n", t2, t0 / t2 / 10);
	else
		printf("Result is wrong.\n");

	return 0;
}